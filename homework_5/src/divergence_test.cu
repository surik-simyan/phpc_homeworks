#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void warmupKernel(int* data)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (true) {
        data[idx] *= 2;
    }
}

__global__ void no_divergence(int* data)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (true) {
        data[idx] *= 2;
    }
}

__global__ void divergence(int* data)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx % 2 == 0)
    {
      data[idx] *= 2;
    }
    else
    {
      data[idx] *= 3;
    }
}

__global__ void small_divergence(int* data)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if ((idx / warpSize) % 2 == 0)
    {
      data[idx] *= 2;
    }
    else
    {
      data[idx] *= 3;
    }
}

int main()
{

    int N = 1000000;
    size_t size = N * sizeof(int);

    int *h_data = (int *)malloc(size);

    for (int i = 0; i < N; i++) {
        h_data[i] = i + 1;
    }

    int *d_data;
    hipMalloc(&d_data, size);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    warmupKernel<<<numBlocks, blockSize>>>(d_data);
    hipDeviceSynchronize();

    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    hipEventRecord(start);
    no_divergence<<<numBlocks, blockSize>>>(d_data);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("No divergence kernel execution time: %.3f ms\n", milliseconds);


    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    hipEventRecord(start);
    divergence<<<numBlocks, blockSize>>>(d_data);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Divergence kernel execution time: %.3f ms\n", milliseconds);


    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    hipEventRecord(start);
    small_divergence<<<numBlocks, blockSize>>>(d_data);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Small divergence kernel execution time: %.3f ms\n", milliseconds);

    free(h_data);
    hipFree(d_data);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
